// compare_cpu_gpu_vector_add.cu
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>

// ---------------- CUDA Kernel ----------------
__global__ void vectorAdd(const int *A, const int *B, int *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

// ---------------- CPU Reference ----------------
void vectorAddCPU(const int *A, const int *B, int *C, int N) {
    for (int i = 0; i < N; i++) C[i] = A[i] + B[i];
}

// ---------------- Benchmark ----------------
void benchmark(int N) {
    size_t size = N * sizeof(int);

    // Host memory
    int *h_A = (int*)malloc(size);
    int *h_B = (int*)malloc(size);
    int *h_C = (int*)malloc(size);
    int *h_ref = (int*)malloc(size);
    for (int i = 0; i < N; i++) { h_A[i] = i; h_B[i] = i * 10; }

    // Device memory
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // CUDA events
    hipEvent_t start, afterH2D, afterKernel, afterD2H;
    hipEventCreate(&start);
    hipEventCreate(&afterH2D);
    hipEventCreate(&afterKernel);
    hipEventCreate(&afterD2H);

    // ---------------- GPU Timing ----------------
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(afterH2D);

    vectorAdd<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C, N);
    hipEventRecord(afterKernel);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(afterD2H);
    hipEventSynchronize(afterD2H);

    float h2d_ms, kernel_ms, d2h_ms, total_ms;
    hipEventElapsedTime(&h2d_ms, start, afterH2D);
    hipEventElapsedTime(&kernel_ms, afterH2D, afterKernel);
    hipEventElapsedTime(&d2h_ms, afterKernel, afterD2H);
    hipEventElapsedTime(&total_ms, start, afterD2H);

    // ---------------- CPU Timing ----------------
    auto t1 = std::chrono::high_resolution_clock::now();
    vectorAddCPU(h_A, h_B, h_ref, N);
    auto t2 = std::chrono::high_resolution_clock::now();
    double cpu_ms = std::chrono::duration<double, std::milli>(t2 - t1).count();

    // ---------------- Print Results ----------------
    printf("N=%d\n", N);
    printf("  CPU      : %.5f ms\n", cpu_ms);
    printf("  GPU Total: %.5f ms (H2D %.5f + Kernel %.5f + D2H %.5f)\n\n",
           total_ms, h2d_ms, kernel_ms, d2h_ms);

    // Validate correctness (optional)
    bool ok = true;
    for (int i = 0; i < 10; i++) {
        if (h_C[i] != h_ref[i]) { ok = false; break; }
    }
    if (!ok) printf("  ❌ Mismatch in results!\n");

    // Cleanup
    free(h_A); free(h_B); free(h_C); free(h_ref);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(afterH2D);
    hipEventDestroy(afterKernel);
    hipEventDestroy(afterD2H);
}

int main() {
    int test_sizes[] = {1024, 10000, 1000000, 10000000};
    for (int N : test_sizes) {
        benchmark(N);
    }
    return 0;
}
