// nvcc -O3 -arch=sm_70 -o matAdd_rt_bench matAdd_rt_bench.cu
// Usage:
//   ./matAdd_rt_bench                # 기본 프리셋들 실행
//   ./matAdd_rt_bench M N            # 지정한 하나의 크기만 실행 (예: 16384 16384)
//   ./matAdd_rt_bench M N --pinned   # Pinned host memory로 전송 대역폭 향상 실험

#include <hip/hip_runtime.h>

#include <cstdio>
#include <vector>
#include <string>
#include <cmath>
#include <cstdlib>

#define CUDA_CHECK(x) do { \
  hipError_t e = (x); \
  if (e != hipSuccess) { \
    fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
    exit(1); \
  } \
} while(0)

__global__ void matAdd(const float* __restrict__ A,
                       const float* __restrict__ B,
                       float* __restrict__ C,
                       int M, int N)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N) {
        int idx = row * N + col; // row-major
        C[idx] = A[idx] + B[idx];
    }
}

struct RunCfg {
    int M, N;
};
struct BlockCfg {
    dim3 block;
    const char* name;
};

static void init_host(float* p, size_t n, float base){
  for(size_t i=0;i<n;++i) p[i] = base + float(i % 1000) * 0.001f;
}

void run_one_case(int M, int N, const BlockCfg& bc, bool usePinned){
  const size_t numel = size_t(M) * size_t(N);
  const size_t bytes = numel * sizeof(float);

  // Host buffers
  float *hA=nullptr, *hB=nullptr, *hC=nullptr;
  if(usePinned){
    CUDA_CHECK(hipHostMalloc(&hA, bytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&hB, bytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&hC, bytes, hipHostMallocDefault));
  }else{
    hA = (float*)malloc(bytes);
    hB = (float*)malloc(bytes);
    hC = (float*)malloc(bytes);
    if(!hA || !hB || !hC){ fprintf(stderr,"host malloc failed\n"); exit(2); }
  }
  init_host(hA, numel, 1.0f);
  init_host(hB, numel, 2.0f);

  // Device buffers
  float *dA=nullptr, *dB=nullptr, *dC=nullptr;
  CUDA_CHECK(hipMalloc(&dA, bytes));
  CUDA_CHECK(hipMalloc(&dB, bytes));
  CUDA_CHECK(hipMalloc(&dC, bytes));

  // Grid calc
  dim3 block = bc.block;
  dim3 grid( (N + block.x - 1) / block.x, (M + block.y - 1) / block.y );

  // Events
  hipEvent_t t0,t1,t2,t3;
  CUDA_CHECK(hipEventCreate(&t0));
  CUDA_CHECK(hipEventCreate(&t1));
  CUDA_CHECK(hipEventCreate(&t2));
  CUDA_CHECK(hipEventCreate(&t3));

  // Record start
  CUDA_CHECK(hipEventRecord(t0));
  // H2D
  CUDA_CHECK(hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipEventRecord(t1));
  // Kernel
  matAdd<<<grid, block>>>(dA, dB, dC, M, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipEventRecord(t2));
  // D2H
  CUDA_CHECK(hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipEventRecord(t3));
  CUDA_CHECK(hipEventSynchronize(t3));

  float ms_H2D=0, ms_K=0, ms_Total=0;
  CUDA_CHECK(hipEventElapsedTime(&ms_H2D, t0, t1));
  CUDA_CHECK(hipEventElapsedTime(&ms_K,   t1, t2));
  CUDA_CHECK(hipEventElapsedTime(&ms_Total, t0, t3));

  // Quick check
  size_t mism=0;
  for(int i=0;i<10;++i){
    size_t idx = (numel/10) * i;
    float ref = hA[idx] + hB[idx];
    if (fabsf(hC[idx]-ref) > 1e-4f) ++mism;
  }

  // Effective bandwidth (GB/s) — read A,B + write C = 3 * bytes
  const double gb_total = (3.0 * double(bytes)) / (1024.0*1024.0*1024.0);
  const double gbps_kernel   = gb_total / (ms_K / 1e3);     // kernel-only
  const double gbps_end2end  = gb_total / (ms_Total / 1e3); // incl H2D+D2H

  printf("M=%d N=%d | Block=%s(%dx%d) Grid=%dx%d | H2D=%.3f ms | Kernel=%.3f ms (%.2f GB/s) | Total=%.3f ms (%.2f GB/s) | mism=%zu | %s\n",
         M, N, bc.name, block.x, block.y, grid.x, grid.y,
         ms_H2D, ms_K, gbps_kernel, ms_Total, gbps_end2end, mism,
         usePinned ? "Pinned" : "Pageable");

  // Cleanup
  CUDA_CHECK(hipEventDestroy(t0));
  CUDA_CHECK(hipEventDestroy(t1));
  CUDA_CHECK(hipEventDestroy(t2));
  CUDA_CHECK(hipEventDestroy(t3));
  CUDA_CHECK(hipFree(dA));
  CUDA_CHECK(hipFree(dB));
  CUDA_CHECK(hipFree(dC));
  if(usePinned){
    CUDA_CHECK(hipHostFree(hA));
    CUDA_CHECK(hipHostFree(hB));
    CUDA_CHECK(hipHostFree(hC));
  }else{
    free(hA); free(hB); free(hC);
  }
}

int main(int argc, char** argv){
  bool usePinned = (argc==4 && std::string(argv[3])=="--pinned");

  std::vector<RunCfg> sizes;
  if (argc >= 3) {
    sizes.push_back({atoi(argv[1]), atoi(argv[2])});
  } else {
    // 기본 프리셋 (필요에 맞게 수정)
    sizes = { {4096,4096}, {8192,8192}, {16384,16384} };
  }

  std::vector<BlockCfg> blocks = {
    {{16,16,1}, "16x16"},
    {{32,8,1 }, "32x8" },
    {{64,4,1 }, "64x4" }
  };

  int dev=0; hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDevice(&dev));
  CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
  printf("[Device] %s | SM=%d | GlobalMem=%.1f GB\n",
         prop.name, prop.multiProcessorCount, prop.totalGlobalMem/ (1024.0*1024.0*1024.0));

  for (auto s : sizes){
    for (auto b : blocks){
      run_one_case(s.M, s.N, b, usePinned);
    }
    printf("----\n");
  }
  return 0;
}
