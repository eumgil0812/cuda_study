// simple_vector.cu
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>

// ----------------- CUDA Kernel -----------------
__global__ void vectorAdd(const int *A, const int *B, int *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

// ----------------- CPU Reference -----------------
void vectorAddCPU(const int *A, const int *B, int *C, int N) {
    for (int i = 0; i < N; ++i) C[i] = A[i] + B[i];
}

int main() {
    const int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(int);

    // Host memory
    int *h_A = (int*)malloc(size);
    int *h_B = (int*)malloc(size);
    int *h_C = (int*)malloc(size);
    int *h_ref = (int*)malloc(size);
    for (int i = 0; i < N; i++) { h_A[i] = i; h_B[i] = i * 10; }

    // Device memory
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Events for CUDA timing
    hipEvent_t start, afterH2D, afterKernel, afterD2H;
    hipEventCreate(&start);
    hipEventCreate(&afterH2D);
    hipEventCreate(&afterKernel);
    hipEventCreate(&afterD2H);

    // ----------------- GPU Timing -----------------
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(afterH2D);

    vectorAdd<<<(N+255)/256, 256>>>(d_A, d_B, d_C, N);
    hipEventRecord(afterKernel);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(afterD2H);
    hipEventSynchronize(afterD2H);

    float h2d_ms, kernel_ms, d2h_ms, total_ms;
    hipEventElapsedTime(&h2d_ms, start, afterH2D);
    hipEventElapsedTime(&kernel_ms, afterH2D, afterKernel);
    hipEventElapsedTime(&d2h_ms, afterKernel, afterD2H);
    hipEventElapsedTime(&total_ms, start, afterD2H);

    // ----------------- CPU Timing -----------------
    auto t1 = std::chrono::high_resolution_clock::now();
    vectorAddCPU(h_A, h_B, h_ref, N);
    auto t2 = std::chrono::high_resolution_clock::now();
    double cpu_ms = std::chrono::duration<double, std::milli>(t2 - t1).count();

    // ----------------- 결과 출력 -----------------
    printf("**** Timer Report ****\n");
    printf("CUDA Total: %.5f ms\n", total_ms);
    printf("Computation (Kernel): %.5f ms\n", kernel_ms);
    printf("Data Trans. Host→Device: %.5f ms\n", h2d_ms);
    printf("Data Trans. Device→Host: %.5f ms\n", d2h_ms);
    printf("VecAdd on CPU: %.5f ms\n", cpu_ms);
    printf("**********************\n");

    // 결과 검증
    bool ok = true;
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_ref[i]) { ok = false; break; }
    }
    printf("Check: %s\n", ok ? "PASS" : "FAIL");

    // Cleanup
    free(h_A); free(h_B); free(h_C); free(h_ref);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(afterH2D);
    hipEventDestroy(afterKernel);
    hipEventDestroy(afterD2H);

    return 0;
}
