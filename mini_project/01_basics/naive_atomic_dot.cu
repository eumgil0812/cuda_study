// naive_atomic_dot.cu (fixed)

#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <random>
#include <cmath>
#include <cstdlib>

#define CUDA_CHECK(expr) do {                             \
  hipError_t _err = (expr);                              \
  if (_err != hipSuccess) {                              \
    fprintf(stderr, "CUDA error %s:%d: %s\n",             \
            __FILE__, __LINE__, hipGetErrorString(_err));\
    exit(1);                                              \
  }                                                       \
} while(0)

__global__ void dot_atomic_naive(const float* A, const float* B, float* out, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        float prod = A[i] * B[i];
        atomicAdd(out, prod); // Single global accumulator
    }
}

int main(int argc, char** argv) {
    int N = 1 << 20;
    size_t bytes = N * sizeof(float);

    // Host
    std::vector<float> A(N), B(N);
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    for (int i = 0; i < N; ++i) { A[i] = dist(rng); B[i] = dist(rng); }

    // Device
    float *dA = nullptr, *dB = nullptr, *dOut = nullptr;
    CUDA_CHECK(hipMalloc(&dA, bytes));
    CUDA_CHECK(hipMalloc(&dB, bytes));
    CUDA_CHECK(hipMalloc(&dOut, sizeof(float)));

    // --- Warm-up: only create context (does not affect the result)
    CUDA_CHECK(hipFree(0));
    CUDA_CHECK(hipDeviceSynchronize());

    // Initialize the result accumulator to zero (important)
    CUDA_CHECK(hipMemset(dOut, 0, sizeof(float)));

    // Copy Host → Device
    CUDA_CHECK(hipMemcpy(dA, A.data(), bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, B.data(), bytes, hipMemcpyHostToDevice));

    // Kernel launch (only once)
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    dot_atomic_naive<<<grid, block>>>(dA, dB, dOut, N);
    CUDA_CHECK(hipGetLastError());      // Check launch errors
    CUDA_CHECK(hipDeviceSynchronize()); // Check runtime errors

    // Retrieve the result
    float gpu = 0.f;
    CUDA_CHECK(hipMemcpy(&gpu, dOut, sizeof(float), hipMemcpyDeviceToHost));

    // CPU reference result (float accumulation)
    float cpu_f = 0.f;
    for (int i = 0; i < N; ++i) cpu_f += A[i] * B[i];

    printf("GPU (atomic) = %.6f\nCPU (float)  = %.6f\nAbs diff = %.6f\n",
      gpu, cpu_f, fabsf(gpu - cpu_f));

    hipFree(dA); hipFree(dB); hipFree(dOut);
    return 0;
}
