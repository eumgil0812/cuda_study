#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    const int N = 10;
    const size_t size = N * sizeof(int);

    int *d_array = NULL;   // Pointer to device memory
    int h_array[N];        // Host memory for verification

    // (1) Allocate device memory
    hipError_t err = hipMalloc((void**)&d_array, size);
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s (%s)\n",
               hipGetErrorName(err), hipGetErrorString(err));
        return -1;
    }

    // (2) Initialize device memory with zeros
    hipMemset(d_array, 0, size);

    // (3) Copy data back from device to host for verification
    hipMemcpy(h_array, d_array, size, hipMemcpyDeviceToHost);

    printf("First %d elements after cudaMemset:\n", N);
    for (int i = 0; i < N; i++) {
        printf("%d ", h_array[i]);
    }
    printf("\n");

    // (4) Free device memory
    hipFree(d_array);

    // (5) Intentional error: request invalid memory size
    int *d_bad = NULL;
    err = hipMalloc((void**)&d_bad, (size_t)-1);
    if (err != hipSuccess) {
        printf("Intentional failure -> %s (%s)\n",
               hipGetErrorName(err), hipGetErrorString(err));
    }

    return 0;
}

